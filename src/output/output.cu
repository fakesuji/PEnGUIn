#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

using namespace std;

#include "parameters.h"
#include "structs.h"
#include "util.h"
#include "forces.cu"

string create_label()
{
	string label;
	if      (ndim==1) label = int_to_string(xres);
	else if (ndim==2) label = int_to_string(xres)+"x"+int_to_string(yres);
	else if (ndim==3) label = int_to_string(xres)+"x"+int_to_string(yres)+"x"+int_to_string(zres);
	
	label += "_h"+int_to_string(1000.0*sc_h);

	if (n_planet>0)
	{
		if (planet_mass>=JupiterMass)
		{
			label += "_"+int_to_string(n_planet)+"p"+int_to_string(planet_mass/JupiterMass)+"J";
		}
		else
		{
			label += "_"+int_to_string(n_planet)+"p"+int_to_string(planet_mass/EarthMass)+"E";
		}
		label += "_e" +int_to_string(planet_ecc*1000.0);
	}

	#if visc_flag == 1
	label += "_a"+int_to_string(log10(ss_alpha)*10.0);
	#elif visc_flag == 2
	label += "_ah"+int_to_string(log10(ss_alpha)*10.0);
	#endif

	#if cool_flag == 1
	label += "_b"+int_to_string(log10(beta_cool)*10.0);
	#elif cool_flag == 2
	label += "_c"+int_to_string(log10(beta_cool)*10.0);
	#elif cool_flag == 3
	label += "_d"+int_to_string(log10(beta_cool)*10.0);
	#endif
	
	#ifdef OrbAdv_flag
	label += "_OA";
	#endif

	#ifdef dust_flag
	label += "_St"+int_to_string(log10(Stokes)*10.0);
	#endif

	#if recon_flag == 0
	label += "_VAN";
	#elif recon_flag == 1
	label += "_MOC";
	#elif recon_flag == 2
	label += "_PEM2";
	#elif recon_flag == 3
	label += "_PEM3";
	#elif recon_flag == 4
	label += "_PPM3";
	#elif recon_flag == 5
	label += "_PEM4";
	#elif recon_flag == 6
	label += "_PPM4";
	#endif

	#if init_flag == 6
	label += "_CFL"+int_to_string(CFL*100.0);
	#endif

	#ifdef rev_flag
	label += "_rev";
	#endif

	//#ifdef ave_flag
	//label += "_ave";
	//#endif

	//label += "_"+int_to_string(ndev)+"dev";
	//label += "_video";

	printf("label %s assigned. \n\n", label.c_str());
 
	return label;
}


void restructure_data(double* data, Grid* G, int data_type)
{
	int ii, jj, kk, glo_idx,loc_idx;

	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		ii = i + G[n].xbgn - xpad;
		jj = j - ypad;
		kk = k - zpad;
		glo_idx = ii + xres*(jj + yres*kk);
		loc_idx = i + G[n].xarr*(j + G[n].yarr*k);
		if      (data_type==0) data[glo_idx] = G[n].C[loc_idx].r;
		else if (data_type==1) data[glo_idx] = G[n].C[loc_idx].p;
		else if (data_type==2) data[glo_idx] = G[n].C[loc_idx].u;
		else if (data_type==3) data[glo_idx] = G[n].C[loc_idx].v;
		else if (data_type==4) data[glo_idx] = G[n].C[loc_idx].w;
		#ifdef dust_flag
		else if (data_type==5) data[glo_idx] = G[n].D[loc_idx].r;
		else if (data_type==6) data[glo_idx] = G[n].D[loc_idx].u;
		else if (data_type==7) data[glo_idx] = G[n].D[loc_idx].v;
		else if (data_type==8) data[glo_idx] = G[n].D[loc_idx].w;
		#endif
		//data[glo_idx] = 0.0;
	}
	return;
}

void write_check_point(ofstream &ofile, double simtime, Grid* G)
{
	size_t memsize= xres*yres*zres*sizeof(double);
	double* tmp = new double[xres*yres*zres];

	ofile.write((char*)&simtime, sizeof(double));

	ofile.write((char*)&G[0].xa[xpad], sizeof(double)*(xres+1));
	#if ndim>1
	ofile.write((char*)&G[0].ya[ypad], sizeof(double)*(yres+1));
	#endif
	#if ndim>2
	ofile.write((char*)&G[0].za[zpad], sizeof(double)*(zres+1));
	#endif

	restructure_data(tmp, G, 0);
	ofile.write((char*)&tmp[0],memsize);

	restructure_data(tmp, G, 1);
	ofile.write((char*)&tmp[0],memsize);

	restructure_data(tmp, G, 2);
	ofile.write((char*)&tmp[0],memsize);

	#if ndim>1
	restructure_data(tmp, G, 3);
	ofile.write((char*)&tmp[0],memsize);
	#endif

	#if ndim>2
	restructure_data(tmp, G, 4);
	ofile.write((char*)&tmp[0],memsize);
	#endif

	#ifdef dust_flag
	restructure_data(tmp, G, 5);
	ofile.write((char*)&tmp[0],memsize);

	restructure_data(tmp, G, 6);
	ofile.write((char*)&tmp[0],memsize);

	#if ndim>1
	restructure_data(tmp, G, 7);
	ofile.write((char*)&tmp[0],memsize);
	#endif

	#if ndim>1
	restructure_data(tmp, G, 8);
	ofile.write((char*)&tmp[0],memsize);
	#endif
	#endif

	delete[] tmp;

	return;
}

double load_grid(Grid* G, string fname)
{
	ifstream start_point;
	open_binary_file(start_point,fname);

	double start_time;
	start_point.read((char*)&start_time, sizeof(double));
	if (ndim>2) 
		start_point.seekg((1+xres+1+yres+1+zres+1)*sizeof(double), ios::beg);
	else if (ndim>1) 
		start_point.seekg((1+xres+1+yres+1)*sizeof(double), ios::beg);
	else
		start_point.seekg((1+xres+1)*sizeof(double), ios::beg);

	
	double tmp;
	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		start_point.read((char*)&tmp, sizeof(double));	
		G[n].write_r(i,j,k,tmp);	
	}

	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		start_point.read((char*)&tmp, sizeof(double));	
		G[n].write_p(i,j,k,tmp);	
	}

	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		start_point.read((char*)&tmp, sizeof(double));	
		G[n].write_u(i,j,k,tmp);	
	}

	if (ndim>1) 
	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		start_point.read((char*)&tmp, sizeof(double));	
		G[n].write_v(i,j,k,tmp);	
	}

	if (ndim>2) 
	for (int k=zpad; k<zres+zpad; k++)
	for (int j=ypad; j<yres+ypad; j++)
	for (int n=0; n<ndev; n++)
	for (int i=xpad; i<G[n].xres+xpad; i++)
	{
		start_point.read((char*)&tmp, sizeof(double));	
		G[n].write_w(i,j,k,tmp);	
	}

	close_output_file(start_point);

	return start_time;
}

__global__ void add_grid(Grid G, Cell* in, Cell* out, double fac)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x + xpad;
	int j = threadIdx.y + blockIdx.y*blockDim.y + ypad;
	int k = threadIdx.z + blockIdx.z*blockDim.z + zpad;

	double xc, yc, zc;

	int ind;

	if (i>=xpad && i<G.xarr-xpad)
	if (j>=ypad && j<G.yarr-ypad)
	if (k>=zpad && k<G.zarr-zpad)
	{		
		ind = G.get_ind(i,j,k);

		out[ind].r += in[ind].r*fac;
		out[ind].p += in[ind].p*fac;
		out[ind].u += in[ind].u*fac;
		out[ind].v += in[ind].v*fac;
		out[ind].w += in[ind].w*fac;
	}

	return;
}

__global__ void zero_grid(Grid G, Cell* in)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x + xpad;
	int j = threadIdx.y + blockIdx.y*blockDim.y + ypad;
	int k = threadIdx.z + blockIdx.z*blockDim.z + zpad;

	double xc, yc, zc;

	int ind;

	if (i>=xpad && i<G.xarr-xpad)
	if (j>=ypad && j<G.yarr-ypad)
	if (k>=zpad && k<G.zarr-zpad)
	{		
		ind = G.get_ind(i,j,k);

		in[ind].r = 0.0;
		in[ind].p = 0.0;
		in[ind].u = 0.0;
		in[ind].v = 0.0;
		in[ind].w = 0.0;
	}

	return;
}

void averaging(Grid* dev, double dt, double t_ave)
{
	int nx, ny, nz;
	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		add_grid<<< dim3(nx/x_xdiv,ny/x_ydiv,nz/x_zdiv), dim3(x_xthd,x_ydiv,x_zdiv), 0, dev[n].stream >>> (dev[n], dev[n].C, dev[n].A, dt/t_ave);
	}
	return;
}

void init_average(Grid* dev)
{
	int nx, ny, nz;
	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		zero_grid<<< dim3(nx/x_xdiv,ny/x_ydiv,nz/x_zdiv), dim3(x_xthd,x_ydiv,x_zdiv), 0, dev[n].stream >>> (dev[n], dev[n].A);
	}
	return;
}

/////////////////////////////////////////////////////////////////

__device__ double cal_torque(Cell gas, body planet, double rad, double azi, double pol, double rad_cyl, double dv)
{
	double mass, acc, torque;

	mass = gas.r * dv;
	acc  = output_gy(rad, azi, pol, planet);
	return rad_cyl * acc * mass;
}

__global__ void get_sum_lv1(double* sum_lv1, double* xa, double* ya, double* za, int nx, int ny, int nz, Cell* C, Dust* CD, body *planets, int m)
{
	extern __shared__ double sm[];
	int i = threadIdx.x;
	int ib = blockIdx.x;
	int ig = i + ib*blockDim.x;
	
	int nmax = (nx*ny*nz + blockDim.x*gridDim.x - 1)/(blockDim.x*gridDim.x);

	int idx, idy, idz;
	int ind;
	double tmp=0.0;
	double rad, azi, pol, rad_cyl;
	double dv;

	for ( int n=0; n<nmax; n++)
	{
		idz = (n+nmax*ig)/(nx*ny);
		idy = (n+nmax*ig - nx*ny*idz)/(nx);
		idx = (n+nmax*ig - nx*idy - nx*ny*idz);
	
		if (idx<nx && idy<ny && idz<nz)
		{
			idx += xpad;
			idy += ypad;
			idz += zpad;
			ind = idx + (nx+2*xpad)*(idy + (ny+2*ypad)*idz);

			#if geomy == 3
			rad = 0.5*(xa[idx+1]+xa[idx]);
			rad_cyl = rad;
			#elif geomy == 4
			rad = 0.5*(xa[idx+1]+xa[idx]);
			rad_cyl = rad * sin(0.5*(za[idz+1]+za[idz]));
			#else	
			rad = 1.0;
			rad_cyl = 1.0;
			#endif
			
			azi = 0.5*(ya[idy+1]+ya[idy]);
			
			#if ndim == 3
			pol = 0.5*(za[idz+1]+za[idz]);
			#else
			pol = 0.0;
			#endif
			
			#if geomy == 3 && ndim == 2
			dv = 0.5*(xa[idx+1]*xa[idx+1] - xa[idx]*xa[idx]);
			#elif geomy == 3 && ndim == 3
			dv = 0.5*(xa[idx+1]*xa[idx+1] - xa[idx]*xa[idx]) * (za[idz+1]-za[idz]);
			#elif geomy == 4
			dv = third*(xa[idx+1]*xa[idx+1]*xa[idx+1] - xa[idx]*xa[idx]*xa[idx]) * (cos(za[idz])-cos(za[idz+1]));
			#else
			dv = (xa[idx+1] - xa[idx]) * (za[idz+1]-za[idz]);
			#endif
			dv *= (ya[idy+1]-ya[idy]);

			tmp += cal_torque(C[ind], planets[m], rad, azi, pol, rad_cyl, dv);
		}
	}
	
	sm[i] = tmp;
	__syncthreads();

	round_reduc_sum(blockDim.x, sm);
	if (i==0) sum_lv1[ib] = sm[i];

	return;
}

__global__ void get_sum_lv2(double* sum, double* sum_lv1)
{
	extern __shared__ double sm[];
	int i = threadIdx.x;

	sm[i] = sum_lv1[i];
	__syncthreads();

	round_reduc_sum(blockDim.x, sm);
	if (i==0) sum[i] = sm[i];

	return;
}

double global_sum(Grid* hst, Grid* dev, int m)
{
	int lv1_size, nx, ny, nz;
	
	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		lv1_size = min(1024,(nx*ny*nz+std_thd-1)/std_thd);
		get_sum_lv1<<< lv1_size, std_thd, std_thd*sizeof(double), dev[n].stream >>>(dev[n].Buff, &dev[n].xa[dev[n].xbgn], dev[n].ya, dev[n].za, nx, ny, nz, dev[n].C, dev[n].CD, dev[n].planets, m);
		get_sum_lv2<<< 1, lv1_size, lv1_size*sizeof(double), dev[n].stream >>>(dev[n].sum, dev[n].Buff);
		hipMemcpyAsync( hst[n].sum, dev[n].sum, sizeof(double), hipMemcpyDeviceToHost, dev[n].stream );
	}
	
	for(int n=0; n<ndev; n++) hipStreamSynchronize(dev[n].stream);

	double tmp = 0.0;
	for (int n=0; n<ndev; n++)
	{
		tmp += *hst[n].sum;
	}

	return tmp;
}
